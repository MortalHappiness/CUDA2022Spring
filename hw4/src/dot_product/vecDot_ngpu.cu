// Includes
#include <hip/hip_runtime.h>
#include <omp.h> // header for OpenMP
#include <stdio.h>
#include <stdlib.h>

// Variables
float *h_A; // host vectors
float *h_B;
float *h_C;

// Functions
void RandomInit(float *, int);

// Device code
__global__ void vecDot(const float *A, const float *B, float *C, int N) {
    extern __shared__ float cache[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int cacheIndex = threadIdx.x;

    if (i < N)
        cache[cacheIndex] = A[i] * B[i];

    __syncthreads();

    // perform parallel reduction, threadsPerBlock must be 2^m

    int ib = blockDim.x / 2;
    while (ib != 0) {
        if (cacheIndex < ib)
            cache[cacheIndex] += cache[cacheIndex + ib];

        __syncthreads();

        ib /= 2;
    }

    if (cacheIndex == 0)
        C[blockIdx.x] = cache[0];
}

// Host code

int main(void) {
    printf("\n");
    printf("Vector Addition with multiple GPUs \n");
    int N = 40960000;
    int NGPU, cpu_thread_id = 0;
    int *Dev;
    long mem = 1024 * 1024 * 1024; // 4 Giga for float data type.

    printf("Enter the number of GPUs: ");
    scanf("%d", &NGPU);
    printf("%d\n", NGPU);
    Dev = (int *)malloc(sizeof(int) * NGPU);

    int numDev = 0;
    printf("GPU device number: ");
    for (int i = 0; i < NGPU; i++) {
        scanf("%d", &Dev[i]);
        printf("%d ", Dev[i]);
        numDev++;
        if (getchar() == '\n')
            break;
    }
    printf("\n");
    if (numDev != NGPU) {
        fprintf(stderr, "Should input %d GPU device numbers\n", NGPU);
        exit(1);
    }

    if (3 * N > mem) {
        printf("The size of these 3 vectors cannot be fitted into 4 Gbyte\n");
        exit(1);
    }
    long size = N * sizeof(float);

    // Set the sizes of threads and blocks
    int threadsPerBlock;
    printf("Enter the number of threads per block: ");
    scanf("%d", &threadsPerBlock);
    printf("%d\n", threadsPerBlock);
    if (threadsPerBlock > 1024) {
        printf("The number of threads per block must be less than 1024 ! \n");
        exit(1);
    }
    int blocksPerGrid =
        (N + threadsPerBlock * NGPU - 1) / (threadsPerBlock * NGPU);
    printf("The number of blocks is %d\n", blocksPerGrid);
    if (blocksPerGrid > 2147483647) {
        printf("The number of blocks must be less than 2147483647 ! \n");
        exit(1);
    }
    int sb = blocksPerGrid * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(sb * NGPU);
    if (!h_A || !h_B || !h_C) {
        printf("!!! Not enough memory.\n");
        exit(1);
    }

    // Initialize input vectors

    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // declare cuda event for timer
    hipEvent_t start, stop;

    float Intime, gputime, Outime;

    omp_set_num_threads(NGPU);

#pragma omp parallel private(cpu_thread_id)
    {
        float *d_A, *d_B, *d_C;
        cpu_thread_id = omp_get_thread_num();
        hipSetDevice(Dev[cpu_thread_id]);

        // start the timer
        if (cpu_thread_id == 0) {
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
        }

        // Allocate vectors in device memory
        hipMalloc((void **)&d_A, size / NGPU);
        hipMalloc((void **)&d_B, size / NGPU);
        hipMalloc((void **)&d_C, sb);

        // Copy vectors from host memory to device memory
        hipMemcpy(d_A, h_A + N / NGPU * cpu_thread_id, size / NGPU,
                   hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B + N / NGPU * cpu_thread_id, size / NGPU,
                   hipMemcpyHostToDevice);
#pragma omp barrier

        // stop the timer
        if (cpu_thread_id == 0) {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&Intime, start, stop);
            printf("Data input time for GPU: %f (ms) \n", Intime);
        }

        // start the timer
        if (cpu_thread_id == 0)
            hipEventRecord(start, 0);

        int sm = threadsPerBlock * sizeof(float);
        vecDot<<<blocksPerGrid, threadsPerBlock, sm>>>(d_A, d_B, d_C, N / NGPU);
        hipDeviceSynchronize();

        // stop the timer

        if (cpu_thread_id == 0) {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&gputime, start, stop);
            printf("Processing time for GPU: %f (ms) \n", gputime);
            printf("GPU Gflops: %f\n", 2 * N / (1000000.0 * gputime));
        }

        // Copy result from device memory to host memory
        // h_C contains the result in host memory

        // start the timer
        if (cpu_thread_id == 0)
            hipEventRecord(start, 0);

        hipMemcpy(h_C + blocksPerGrid * cpu_thread_id, d_C, sb,
                   hipMemcpyDeviceToHost);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        // stop the timer

        if (cpu_thread_id == 0) {
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&Outime, start, stop);
            printf("Data output time for GPU: %f (ms) \n", Outime);
        }
    }

    double h_G = 0.0;
    for (int i = 0; i < blocksPerGrid * NGPU; i++)
        h_G += (double)h_C[i];

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n", gputime_tot);

    // start the timer
    hipEventRecord(start, 0);

    // to compute the reference solution

    double h_D = 0.0;
    for (int i = 0; i < N; i++)
        h_D += (double)h_A[i] * h_B[i];

    // stop the timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime(&cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n", cputime);
    printf("CPU Gflops: %f\n", 2 * N / (1000000.0 * cputime));
    printf("Speed up of GPU = %f\n", cputime / gputime_tot);

    // Destroy timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result
    printf("Check result:\n");
    double diff = abs((h_D - h_G) / h_D);
    printf("|(h_G - h_D)/h_D|=%20.15e\n", diff);
    printf("h_G =%20.15e\n", h_G);
    printf("h_D =%20.15e\n", h_D);

    for (int i = 0; i < NGPU; i++) {
        hipSetDevice(i);
        hipDeviceReset();
    }

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

// Allocates an array with random float entries.
void RandomInit(float *data, int n) {
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
